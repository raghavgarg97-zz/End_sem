
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

int seed;

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);


__global__ void XOR(int *da,int cskip, int num)
{
	
	
	long long int thread_no=blockDim.x * blockIdx.x + threadIdx.x;
	long long int start= thread_no*cskip;
	if(start>=num)return;	
	long long int end=start+cskip-1;
	if(end>=num){end=num-1;}

	
	if(start==end)return;
	if(da[end]!=-1){
		if(thread_no%2!=0){da[end]=da[end]^da[start];da[start]=-1;}
		else {da[start]=da[start]^da[end];da[end]=-1;}
	}
	else {
		if(thread_no%2!=0){da[end]=da[start];da[start]=-1;}
		else {da[end]=-1;}
		

	}
	
	
 
}


int main(int argc,char** argv){

int *ar,num;

if(argc!=3){
		printf("Invalid number of Arguments");
		exit(-1);
}

num=atoi(argv[1]);
seed=atoi(argv[2]);

if(num<=0){
printf("Invalid Number");
		exit(-1);
}

ar=(int*)malloc(num*sizeof(int));
 if(!ar){
          perror("malloc");
          exit(-1);
    }

srand(seed);
for(int i=0;i<num;i++){
	ar[i]=random();
}


int skip=2;
int size=num*sizeof(int);
int xor_output;

int*dA;
hipMalloc(&dA,size);
CUDA_ERROR_EXIT("hipMalloc");

//int*cskip;
//cudaMalloc(&cskip,sizeof(int));
//CUDA_ERROR_EXIT("cudaMalloc");

hipMemcpy(dA, ar, size, hipMemcpyHostToDevice);
CUDA_ERROR_EXIT("memcpy1");

//cudaMemcpy(&xor_output,&dA[0],sizeof(int), cudaMemcpyDeviceToHost);
//printf("%d\n",xor_output);

//cudaMemcpy(cskip,&skip, sizeof(int), cudaMemcpyHostToDevice);
//CUDA_ERROR_EXIT("memcpy1");


while(skip/2<num){
int num_threads=num/skip;
if(num%skip)num_threads++;
int num_blocks=num_threads/1024;
if(num_threads%1024)num_blocks++;
XOR<<<num_blocks, 1024 >>>(dA,skip,num);
CUDA_ERROR_EXIT("kernel invocation");
skip*=2;
}

hipMemcpy(&xor_output,&dA[0],sizeof(int), hipMemcpyDeviceToHost);
CUDA_ERROR_EXIT("memcpy2");

printf("%d\n",xor_output);

hipFree(dA);
free(ar);

return 0;
}
